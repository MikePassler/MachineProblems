#include "hip/hip_runtime.h"
//Michael Passler (20167458)
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> // For rand() and srand()
#include <time.h> // For time()
#include <omp.h>

__global__ void matrixAdd(float* A, float* B, float* C, int matrixDim) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < matrixDim && j < matrixDim) {
		int index = i * matrixDim + j;
		C[index] = A[index] + B[index];
	}
}

__global__ void matrixAddCol(float* A, float* B, float* C, int matrixDim) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < matrixDim) {
		for (int row = 0; row < matrixDim; row++) {
			int index = row * matrixDim + col;
			C[index] = A[index] + B[index];
		}
	}
	
}

__global__ void matrixAddRow(float* A, float *B, float* C, int matrixDim) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row < matrixDim) {
		for (int col = 0; col < matrixDim; col++) {
			int index = row * matrixDim + col;
			C[index] = A[index] + B[index];
		}
	}
}

void populateMatrix(float *X, int matrixDim) {
	srand(time(NULL));

	// Populate the matrix with random floating point values
	for (int i = 0; i < matrixDim; i++) {
		for (int j = 0; j < matrixDim; j++) {
			X[i*matrixDim + j] = (float)rand() / RAND_MAX; // Generate a random floating point value between 0 and 1
		}
	}
}

int main()
{
	
	float *h_A;
	float *h_B;
	float *h_C;

	int matrixDim = 125;
	int matrixSize = matrixDim*matrixDim;
	int bytes = matrixSize*sizeof(float);

	//printf("bytes specified");
	//hipGetLastError();
	hipHostMalloc((void**)&h_A, bytes);
	hipHostMalloc((void**)&h_B, bytes);
	hipHostMalloc((void**)&h_C, bytes);
	//printf("host mem allocated");

	float *d_A;
	float *d_B;
	float *d_C;

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);
	//printf("device mem allocated");

	//fill h_A and h_B and h_C
	populateMatrix(h_A, matrixDim);
	populateMatrix(h_B, matrixDim);
	for (int i = 0; i < matrixDim; i++) {
		for (int j = 0; j < matrixDim; j++) {
			h_C[i*matrixDim + j] = 0; 
		}
	}

	hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_C, h_C, bytes, hipMemcpyHostToDevice);
	//printf("device mem transfered");

	// Initialize the CUDA device
	hipSetDevice(0);

	// Define the thread block dimensions
	//dim3 threads(16, 16, 1);
	//dim3 threads(16, 1, 1);
	dim3 threads(1, 16, 1);



	int threadsPerBlock = threads.x * threads.y;


	int remainder = matrixDim / threadsPerBlock;
	int numBlocks = matrixDim / threadsPerBlock;

	if (remainder) {
		numBlocks = numBlocks + 1;
	}

	

	// Launch the kernel function with the specified thread block and grid dimensions
	int x = threads.x;
	int y = threads.y;
	//printf("%d", x);
	//printf("%d", y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Wait for all CUDA threads to finish
	hipDeviceSynchronize();

	float milliseconds = 0.0f;
	// Record the start event
	hipEventRecord(start);

	if (threads.x == 16 && threads.y == 16) {
		dim3 blocks(numBlocks, numBlocks, 1);
		matrixAdd << <blocks, threads >> >(d_A, d_B, d_C, matrixDim);
	}

	else if (threads.x == 16 && threads.y == 1) {
		dim3 blocks(numBlocks, 1, 1);
		matrixAddCol << <blocks, threads >> >(d_A, d_B, d_C, matrixDim);
	}

	else {
		dim3 blocks(1, numBlocks, 1);
		matrixAddRow << <blocks, threads >> >(d_A, d_B, d_C, matrixDim);
	}

	// Record the stop event
	hipEventRecord(stop);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Calculate the elapsed time in milliseconds
	
	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the elapsed time
	printf("Time taken by GPU: %.2f ms\n", milliseconds);

	hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost);

	//now time the cpu addition and compare.

	float *CPU;
	CPU = (float*)malloc(bytes);
	int flag = 0;
	float begin, finish;

	begin = omp_get_wtime();
	//FULL ADD and row add test
	
	for (int i = 0; i < matrixDim; i++) {
		for (int k = 0; k < matrixDim; k++) {
			int index = i*matrixDim + k;
			CPU[index] = h_A[index] + h_B[index];
			if (CPU[index] != h_C[index]) {
				flag = 1;
			}
		}
	}
	

	//COL ADD
	/*
	for (int i = 0; i < matrixDim; i++) {
		for (int j = 0; j < matrixDim; j++) {
			int index = j*matrixDim + i;
			CPU[index] = h_A[index] + h_B[index];
			if (CPU[index] != h_C[index]) {
				flag = 1;
			}
		}
	}
	*/
	if (flag == 1) { printf("failed\n"); }
	else printf("succeeded\n");

	finish = omp_get_wtime();
	float deltaT = (finish - begin);
	printf("CPU time: %f\n", (deltaT)*1000);


	




	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("DONE\n");
	
	return 0;


}


