#include "hip/hip_runtime.h"
//Michael Passler (20167458)
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h> // For rand() and srand()
#include <time.h> // For time()
#include <omp.h>

__global__ void matrixMul(float *A, float *B, float *C, int matrixSize) {
	// Compute the row and column indices of the element
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Perform matrix multiplication only for indices within the matrix dimensions
	if (row < matrixSize && col < matrixSize) {
		float sum = 0.0f;
		for (int k = 0; k < matrixSize; k++) {
			float a = A[row * matrixSize + k];
			float b = B[k * matrixSize + col];
			sum += a * b;
		}
		C[row * matrixSize + col] = sum;
	}

}

void populateMatrix(float *X, int matrixDim) {
	srand(time(NULL));

	// Populate the matrix with random floating point values
	for (int i = 0; i < matrixDim; i++) {
		for (int j = 0; j < matrixDim; j++) {
			X[i*matrixDim + j] = (float)rand() / RAND_MAX; // Generate a random floating point value between 0 and 1
		}
	}
}

int main()
{

	float *h_A;
	float *h_B;
	float *h_C;

	int matrixDim = 125;
	int matrixSize = matrixDim*matrixDim;
	int bytes = matrixSize*sizeof(float);

	//printf("bytes specified");
	//hipGetLastError();
	hipHostMalloc((void**)&h_A, bytes);
	hipHostMalloc((void**)&h_B, bytes);
	hipHostMalloc((void**)&h_C, bytes);
	//printf("host mem allocated");

	float *d_A;
	float *d_B;
	float *d_C;

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);
	//printf("device mem allocated");

	//fill h_A and h_B and h_C
	populateMatrix(h_A, matrixDim);
	populateMatrix(h_B, matrixDim);
	for (int i = 0; i < matrixDim; i++) {
		for (int j = 0; j < matrixDim; j++) {
			h_C[i*matrixDim + j] = 0;
		}
	}

	//PART 1 -------------------------------------------------------------------------------------------------
	/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Wait for all CUDA threads to finish
	hipDeviceSynchronize();

	float milliseconds = 0.0f;
	// Record the start event
	hipEventRecord(start);

	hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);

	// Initialize the CUDA device
	hipSetDevice(0);

	// Record the stop event
	hipEventRecord(stop);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Calculate the elapsed time in milliseconds

	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the elapsed time
	printf("GPU host to device: %.2f ms\n", milliseconds);

	//hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost);

	//now time the cpu addition and compare.
	*/
	//PART1A-------------------------------------------------------------------------------------------------------------
	/*
	hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);

	// Initialize the CUDA device
	hipSetDevice(0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Wait for all CUDA threads to finish
	hipDeviceSynchronize();

	float milliseconds = 0.0f;
	// Record the start event
	hipEventRecord(start);

	hipMemcpyAsync(h_A, d_A, bytes, hipMemcpyDeviceToHost);
	hipMemcpyAsync(h_B, d_B, bytes, hipMemcpyDeviceToHost);

	// Initialize the CUDA device
	hipSetDevice(0);

	// Record the stop event
	hipEventRecord(stop);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Calculate the elapsed time in milliseconds

	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the elapsed time
	printf("GPU device to host: %.2f ms\n", milliseconds);
	*/
//PART 2----------------------------------------------------------------------------------------------------------

	hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);
	dim3 threadsPerBlock(1, matrixSize, 1);
	dim3 numBlocks(1, 1, 1);
	// Initialize the CUDA device
	hipSetDevice(0);
	float milliseconds = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Wait for all CUDA threads to finish
	hipDeviceSynchronize();

	matrixMul << <numBlocks, threadsPerBlock >> >(d_A, d_B, d_C, matrixSize);


	// Record the stop event
	hipEventRecord(stop);

	// Wait for the stop event to complete
	hipEventSynchronize(stop);

	// Calculate the elapsed time in milliseconds

	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the elapsed time
	printf("Time taken by GPU: %.2f ms\n", milliseconds);
	
	hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost);
	
	/*
	float *CPU;
	CPU = (float*)malloc(bytes);
	int flag = 0;
	float begin, finish;

	begin = omp_get_wtime();


	if (flag == 1) { printf("failed\n"); }
	else printf("succeeded\n");

	finish = omp_get_wtime();
	float deltaT = (finish - begin);
	printf("CPU time: %f\n", (deltaT)* 1000);




	*/


	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("DONE\n");

	return 0;


}
